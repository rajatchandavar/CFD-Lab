#include "hip/hip_runtime.h"
#include "CUDA_solver.cuh"

#define at(var, i, j) var[ (j) * (*size_x) + (i)]

__device__ double interpolate(double *A, int i, int j, int i_offset, int j_offset, int *size_x) {
    double result =( at(A, i, j) + at(A, i + i_offset, j + j_offset)) / 2;
    return result;
}

__device__ double diffusion(double *A, int i, int j, double dx, double dy, int *size_x) {
    double result = (at(A, i + 1, j) - 2.0 * at(A, i, j) + at(A, i - 1, j)) / (dx * dx) +
                    (at(A, i, j + 1) - 2.0 * at(A, i, j) + at(A, i, j - 1)) / (dy * dy);

    return result;
}

__device__ double convection_Tu(double *T, double *U, int i, int j, double dx, double dy, double gamma, int *size_x)
{
    double result;
    result = 1/dx * ( at(U, i, j) * interpolate(T,i,j,1,0, size_x) - at(U, i - 1,j) * interpolate(T,i-1,j,1,0, size_x)) + 
                        gamma/dx * ( fabsf(at(U, i, j)) * (at(T, i, j) - at(T, i + 1, j)) / 2 - fabsf(at(U, i - 1,j)) * (at(T, i - 1, j) - at(T, i, j)) / 2 );
    return result;
}

__device__ double convection_Tv(double *T, double *V, int i, int j, double dx, double dy, double gamma, int *size_x)
{
    double result;
    result = 1/dy * ( at(V, i, j) * interpolate(T,i,j,0,1, size_x) - at(V, i,j - 1) * interpolate(T,i,j - 1,0,1, size_x)) + 
                        gamma/dy * ( fabsf(at(V, i, j)) * (at(T, i, j) - at(T, i, j + 1)) / 2 - fabsf(at(V, i,j - 1)) * (at(T, i, j - 1) - at(T, i, j)) / 2 );
    return result;
}

dim3 CUDA_solver::get_num_blocks(int size) { return (size + BLOCK_SIZE - 1) / BLOCK_SIZE; }

dim3 CUDA_solver::get_num_blocks_2d(int size_x, int size_y){
    return (dim3((size_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (size_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y));
}

__global__ void calc_T_kernel(double * T, double *T_temp, double *U, double *V, double *dx, double *dy, double *dt, double *alpha, double *gamma, int *size_x, int *size_y){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *size_x && j < *size_y)
        at(T,i,j) = (*dt) * (*alpha * diffusion(T_temp,i,j, *dx, *dy, size_x) - convection_Tu(T_temp,U,i,j,*dx, *dy, *gamma, size_x) - convection_Tv(T_temp,V,i,j, *dx, *dy, *gamma, size_x)) + at(T_temp, i,j);
    
}

CUDA_solver::CUDA_solver(Fields &field, Grid &grid){

    block_size = dim3(BLOCK_SIZE);
    block_size_2d = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    domain_size = (grid.domain().domain_size_x + 2) * (grid.domain().domain_size_y + 2);
    hipMalloc(&geometry_file, domain_size * sizeof(int));

    grid_size = grid.imaxb() * grid.jmaxb();
    grid_size_x = grid.imaxb();
    grid_size_y = grid.jmaxb();
    hipMalloc((void **)&T, grid_size * sizeof(double));
    hipMalloc((void **)&T_temp, grid_size * sizeof(double));
    hipMalloc((void **)&U, grid_size * sizeof(double));
    hipMalloc((void **)&V, grid_size * sizeof(double));

    hipMalloc((void **)&dx, sizeof(double));
    hipMalloc((void **)&dy, sizeof(double));
    hipMalloc((void **)&dt, sizeof(double));
    hipMalloc((void **)&alpha, sizeof(double));
    hipMalloc((void **)&gamma, sizeof(double));
    hipMalloc((void **)&size_x, sizeof(double));
    hipMalloc((void **)&size_y, sizeof(double));
}

void CUDA_solver::pre_process(Fields &field, Grid &grid, Discretization &discretization){
    
    hipMemcpy(geometry_file, grid.get_geometry_data().data(), domain_size * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(T, field.t_matrix().data(), grid_size * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(U, field.u_matrix().data(), grid_size * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(V, field.v_matrix().data(), grid_size * sizeof(double),hipMemcpyHostToDevice);


    double var = grid.dx();
    hipMemcpy(dx, &var, sizeof(double),hipMemcpyHostToDevice);
    var = grid.dy();
    hipMemcpy(dy, &var, sizeof(double),hipMemcpyHostToDevice);
    var = field.dt();
    hipMemcpy(dt, &var, sizeof(double),hipMemcpyHostToDevice);
    var = field.get_alpha();
    hipMemcpy(alpha, &var, sizeof(double),hipMemcpyHostToDevice);
    var = discretization.get_gamma();
    hipMemcpy(gamma, &var, sizeof(double),hipMemcpyHostToDevice);

    hipMemcpy(size_x, &grid_size_x, sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(size_y, &grid_size_y, sizeof(int),hipMemcpyHostToDevice);

}

void CUDA_solver::calc_T(){
    hipMemcpy(T_temp, T, grid_size * sizeof(double), hipMemcpyDeviceToDevice);
    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);
    calc_T_kernel<<<num_blocks_2d, block_size_2d>>>(T, T_temp, U, V, dx, dy, dt, alpha, gamma, size_x, size_y);
}

void CUDA_solver::apply_boundary(){

}

void CUDA_solver::calc_fluxes(){
    
}

void CUDA_solver::calc_rs(){
    
}

void CUDA_solver::post_process(Fields &field){
    hipMemcpy((void *)field.t_matrix().data(), T, grid_size * sizeof(double), hipMemcpyDeviceToHost);
}

CUDA_solver::~CUDA_solver(){
    hipFree(geometry_file);
    hipFree(T);
    hipFree(T_temp);
    hipFree(dx);
    hipFree(dy);
    hipFree(dt);
    hipFree(gamma);
    hipFree(alpha);
}
