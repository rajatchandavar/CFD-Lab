#include "hip/hip_runtime.h"
#include "CUDA_solver.cuh"
#include <cmath>
#include <iostream>
#include <iomanip> 
#include <math.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif
#define at(var, i, j) var[(j) * (*gpu_size_x) + (i)]

#define check_bound_north(j) ((j + 1) < *gpu_size_y)
#define check_bound_south(j) ((j - 1) >= 0)
#define check_bound_east(i) ((i + 1) < *gpu_size_x)
#define check_bound_west(i) ((i - 1) >=0)

__device__ dtype interpolate(dtype *A, int i, int j, int i_offset, int j_offset, int *gpu_size_x) {
    dtype result = (at(A, i, j) + at(A, i + i_offset, j + j_offset)) / 2;
    return result;
}

__device__ dtype diffusion(dtype *A, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {
    dtype result = (at(A, i + 1, j) - 2.0 * at(A, i, j) + at(A, i - 1, j)) / (gpu_dx * gpu_dx) +
                    (at(A, i, j + 1) - 2.0 * at(A, i, j) + at(A, i, j - 1)) / (gpu_dy * gpu_dy);

    return result;
}

__device__ dtype convection_u(dtype *gpu_U, dtype *gpu_V, dtype gpu_gamma, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {

    dtype t1 = interpolate(gpu_U, i, j, 1, 0, gpu_size_x);
    dtype t2 = interpolate(gpu_U, i, j, -1, 0, gpu_size_x);
    dtype du2_dx = 1 / gpu_dx * ((t1 * t1) - (t2 * t2)) + gpu_gamma / gpu_dx *\
            ((fabsf(interpolate(gpu_U, i, j, 1, 0, gpu_size_x)) * (at(gpu_U, i, j) - at(gpu_U, i + 1, j)) / 2) -\
             fabsf(interpolate(gpu_U, i, j, -1, 0, gpu_size_x)) * (at(gpu_U, i - 1, j) - at(gpu_U, i, j)) / 2);
    dtype duv_dy = 1 / gpu_dy * (((interpolate(gpu_V, i, j, 1, 0, gpu_size_x)) * (interpolate(gpu_U, i, j, 0, 1, gpu_size_x))) -\
                         ((interpolate(gpu_V, i, j - 1, 1, 0, gpu_size_x)) * (interpolate(gpu_U, i, j, 0, -1, gpu_size_x)))) +\
                    gpu_gamma / gpu_dy *\
                        ((fabsf(interpolate(gpu_V, i, j, 1, 0, gpu_size_x)) * (at(gpu_U, i, j) - at(gpu_U, i, j + 1)) / 2) -\
                         (fabsf(interpolate(gpu_V, i, j - 1, 1, 0, gpu_size_x)) * (at(gpu_U, i, j - 1) - at(gpu_U, i, j)) / 2));

    dtype result = du2_dx + duv_dy;
    return result;
}

__device__ dtype convection_v(dtype *gpu_U, dtype *gpu_V, dtype gpu_gamma, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {

    dtype t1 = interpolate(gpu_V, i, j, 0, 1, gpu_size_x);
    dtype t2 = interpolate(gpu_V, i, j - 1, 0, 1, gpu_size_x);
    dtype dv2_dy = 1 / gpu_dy * ((t1 * t1) - (t2 * t2)) + gpu_gamma / gpu_dy *\
            ((fabsf(interpolate(gpu_V, i, j, 0, 1, gpu_size_x)) * (at(gpu_V, i, j) - at(gpu_V, i, j + 1)) / 2) -\
             fabsf(interpolate(gpu_V, i, j - 1, 0, 1, gpu_size_x)) * (at(gpu_V, i, j - 1) - at(gpu_V, i, j)) / 2);
    dtype duv_dx = 1 / gpu_dx *\
                        (((interpolate(gpu_U, i, j, 0, 1, gpu_size_x)) * (interpolate(gpu_V, i, j, 1, 0, gpu_size_x))) -\
                         ((interpolate(gpu_U, i - 1, j, 0, 1, gpu_size_x)) * (interpolate(gpu_V, i - 1, j, 1, 0, gpu_size_x)))) +\
                    gpu_gamma / gpu_dx *\
                        ((fabsf(interpolate(gpu_U, i, j, 0, 1, gpu_size_x)) * (at(gpu_V, i, j) - at(gpu_V, i + 1, j)) / 2) -\
                         (fabsf(interpolate(gpu_U, i - 1, j, 0, 1, gpu_size_x)) * (at(gpu_V, i - 1, j) - at(gpu_V, i, j)) / 2));

    dtype result = dv2_dy + duv_dx;
    return result;
}

__device__ dtype convection_Tu(dtype *gpu_T, dtype *gpu_U, int i, int j, dtype gpu_dx, dtype gpu_dy, dtype gpu_gamma, int *gpu_size_x) {
    dtype result;
    result = 1 / gpu_dx * (at(gpu_U, i, j) * interpolate(gpu_T, i, j, 1, 0, gpu_size_x) - at(gpu_U, i - 1, j) * interpolate(gpu_T, i - 1, j, 1, 0, gpu_size_x)) +
             gpu_gamma / gpu_dx * (fabsf(at(gpu_U, i, j)) * (at(gpu_T, i, j) - at(gpu_T, i + 1, j)) / 2 - fabsf(at(gpu_U, i - 1, j)) * (at(gpu_T, i - 1, j) - at(gpu_T, i, j)) / 2);
    return result;
}

__device__ dtype convection_Tv(dtype *gpu_T, dtype *gpu_V, int i, int j, dtype gpu_dx, dtype gpu_dy, dtype gpu_gamma, int *gpu_size_x) {
    dtype result;
    result = 1 / gpu_dy * (at(gpu_V, i, j) * interpolate(gpu_T, i, j, 0, 1, gpu_size_x) - at(gpu_V, i, j - 1) * interpolate(gpu_T, i, j - 1, 0, 1, gpu_size_x)) +
             gpu_gamma / gpu_dy * (fabsf(at(gpu_V, i, j)) * (at(gpu_T, i, j) - at(gpu_T, i, j + 1)) / 2 - fabsf(at(gpu_V, i, j - 1)) * (at(gpu_T, i, j - 1) - at(gpu_T, i, j)) / 2);
    return result;
}

__device__ dtype sor_helper(dtype *gpu_P, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {
    dtype result = (at(gpu_P,i + 1, j) + at(gpu_P,i - 1, j)) / (gpu_dx * gpu_dx) + (at(gpu_P,i, j + 1) + at(gpu_P,i, j - 1)) / (gpu_dy * gpu_dy);
    return result;
}

__device__ dtype laplacian(dtype *gpu_P, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {
    dtype result = (at(gpu_P,i + 1, j) - 2.0 * at(gpu_P,i, j) + at(gpu_P,i - 1, j)) / (gpu_dx * gpu_dx) +
                   (at(gpu_P,i, j + 1) - 2.0 * at(gpu_P,i, j) + at(gpu_P,i, j - 1)) / (gpu_dy * gpu_dy);
    return result;
}

dim3 CUDA_solver::get_num_blocks(int size) { return (size + BLOCK_SIZE - 1) / BLOCK_SIZE; }

dim3 CUDA_solver::get_num_blocks_2d(int gpu_size_x, int gpu_size_y) {
    return (dim3((gpu_size_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (gpu_size_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y));
}

__global__ void FixedWallBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, dtype *gpu_T, int *gpu_geometry_data,
int *gpu_fluid_id, int *gpu_moving_wall_id, int *gpu_fixed_wall_id, int *gpu_inflow_id, int *gpu_outflow_id, int *gpu_adiabatic_id, int *gpu_hot_id,
int *gpu_cold_id, dtype *gpu_wall_temp_a, dtype *gpu_wall_temp_h, dtype *gpu_wall_temp_c, bool *gpu_isHeatTransfer, int *gpu_size_x, int
*gpu_size_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && (at(gpu_geometry_data, i, j) == 3 || at(gpu_geometry_data, i, j) == 5 || at(gpu_geometry_data, i, j) == 6 || at(gpu_geometry_data, i, j) == 7)) {
        // obstacles B_NE (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the North and East directions 

        if(check_bound_north(j) && check_bound_east(i) && at(gpu_geometry_data,i,j+1)==*gpu_fluid_id && at(gpu_geometry_data,i+1,j)==*gpu_fluid_id) {
            at(gpu_U, i, j) = 0.0;
            at(gpu_U, i - 1, j) = -at(gpu_U, i - 1, j + 1);
            at(gpu_V, i, j) = 0.0;
            at(gpu_V, i, j - 1) = -at(gpu_V, i + 1, j - 1);
            at(gpu_P, i, j) = (at(gpu_P, i, j + 1) + at(gpu_P, i + 1, j))/2;

            if(*gpu_isHeatTransfer==1) {
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T, i, j) = (at(gpu_T, i + 1, j) + at(gpu_T, i, j + 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j + 1) + at(gpu_T,i + 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j + 1) + at(gpu_T,i + 1, j) )/2;
            }
        }

        // obstacles B_SE (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the South and East directions 

        else if(check_bound_south(j) && check_bound_east(i) && at(gpu_geometry_data,i,j-1)==*gpu_fluid_id&& at(gpu_geometry_data,i+1,j)==*gpu_fluid_id) {

            at(gpu_U, i, j) = 0.0;
            at(gpu_U, i - 1, j) = -at(gpu_U, i - 1, j - 1);
            at(gpu_V, i, j - 1) = 0.0;
            at(gpu_V, i, j) = -at(gpu_V, i + 1, j);
            at(gpu_P,i, j) = (at(gpu_P,i + 1, j) + at(gpu_P,i, j - 1))/2;

            if(*gpu_isHeatTransfer==1) {
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = (at(gpu_T, i + 1, j) + at(gpu_T, i, j - 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T, i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j - 1) + at(gpu_T,i + 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j - 1) + at(gpu_T,i + 1, j) )/2;
            }

        }

        // obstacle B_NW (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the North and West directions 
            
        else if(check_bound_north(j) && check_bound_west(i) && at(gpu_geometry_data,i,j+1)==*gpu_fluid_id && at(gpu_geometry_data,i-1,j)==*gpu_fluid_id) {

            at(gpu_U,i - 1, j) = 0.0;
            at(gpu_U,i, j) = -at(gpu_U,i, j + 1);
            at(gpu_V,i, j) = 0.0;
            at(gpu_V,i, j - 1) = -at(gpu_V,i - 1, j - 1);
            at(gpu_P,i,j) = (at(gpu_P,i - 1, j) + at(gpu_P,i, j + 1))/2;

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = (at(gpu_T,i - 1, j) + at(gpu_T,i, j + 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j + 1) + at(gpu_T,i - 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j + 1) + at(gpu_T,i - 1, j) )/2;
            }

        }

        // obstacle B_SW (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the South and West directions 

        else if(check_bound_south(j) && check_bound_west(i) && at(gpu_geometry_data,i,j-1)==*gpu_fluid_id&& at(gpu_geometry_data,i-1,j)==*gpu_fluid_id){
            at(gpu_U,i - 1, j) = 0.0;
            at(gpu_U,i, j) = at(gpu_U,i, j - 1);
            at(gpu_V,i, j - 1) = 0.0;
            at(gpu_V,i, j) = -at(gpu_V,i - 1, j);
            at(gpu_P,i, j) = (at(gpu_P,i - 1, j) + at(gpu_P,i, j - 1))/2;
           
            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                   at(gpu_T,i, j) = (at(gpu_T,i - 1, j) + at(gpu_T,i, j - 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j - 1) + at(gpu_T,i - 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j - 1) + at(gpu_T,i - 1, j) )/2;
            }
            
        }

        // Bottom Wall B_N (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the North direction

        else if(check_bound_north(j) && at(gpu_geometry_data,i,j+1)==*gpu_fluid_id){
            at(gpu_U,i, j) = -at(gpu_U,i, j + 1);
            at(gpu_V,i, j) = 0.0;
            at(gpu_P,i, j) = at(gpu_P,i, j + 1);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i, j + 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - at(gpu_T,i, j + 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - at(gpu_T,i, j + 1);
            }
        }

        // Top Wall B_S (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the South direction

        else if(check_bound_south(j) && at(gpu_geometry_data,i,j-1)==*gpu_fluid_id){

            at(gpu_U,i, j) = -at(gpu_U,i, j - 1);
            at(gpu_V,i, j) = 0.0;
            at(gpu_P,i, j) = at(gpu_P,i, j - 1);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i, j - 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2 * (*gpu_wall_temp_h) - at(gpu_T,i, j - 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2 * (*gpu_wall_temp_c) - at(gpu_T,i, j - 1);
            }
        }

        // Left Wall B_E (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the East direction

        else if(check_bound_east(i) && at(gpu_geometry_data,i+1,j)==*gpu_fluid_id){
            at(gpu_U,i, j) = 0.0;
            at(gpu_V,i, j) = -at(gpu_V,i + 1, j);
            at(gpu_P,i, j) = at(gpu_P,i + 1, j);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i + 1, j);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id) 
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - at(gpu_T,i + 1, j);        
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - at(gpu_T,i + 1, j);
            }
        }

        
        /***********************************************************************************************
        * Right Wall B_W (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on the West direction *
        ***********************************************************************************************/

        else if(check_bound_west(i) && at(gpu_geometry_data,i-1,j)==0){
            //Since u grid is staggered, the u velocity of cells to left of ghost layer should be set to 0.
            at(gpu_U,i - 1, j) = 0.0; 
            at(gpu_V,i, j) = -at(gpu_V,i - 1, j);
            at(gpu_P,i, j) = at(gpu_P,i - 1, j);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i - 1, j);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - at(gpu_T,i - 1, j);
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - at(gpu_T,i - 1, j);
            }
        }
    }
}

__global__ void MovingWallBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_moving_wall_id, dtype *gpu_wall_velocity, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_moving_wall_id && check_bound_south(j)) {       
        at(gpu_U,i, j) = 2*(*gpu_wall_velocity)- at(gpu_U,i, j-1);
        //Since v grid is staggered, the v velocity of cells to below of ghost layer should be set to 0.
        at(gpu_V,i,j - 1) = 0.0;
        at(gpu_P,i,j) = at(gpu_P,i, j-1);
    }
}

__global__ void InFlowBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_inflow_id, dtype *gpu_UIN, dtype *gpu_VIN, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (check_bound_east(i) && i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_inflow_id) {
        at(gpu_U,i,j) = (*gpu_UIN);
        at(gpu_V,i,j) = 2*(*gpu_VIN) - at(gpu_V,i + 1, j);
        at(gpu_P,i,j) = at(gpu_P,i + 1, j);
    }
}

__global__ void OutFlowBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_outflow_id, dtype *gpu_POUT, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (check_bound_west(i) && i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_outflow_id) {
            at(gpu_U,i,j) = at(gpu_U,i - 1,j);
            at(gpu_V,i,j) = at(gpu_V,i - 1,j);
            at(gpu_P,i,j) = 2*(*gpu_POUT) - at(gpu_P,i - 1, j);
    }
}

__global__ void calc_T_kernel(dtype *gpu_T, dtype *gpu_T_temp, dtype *gpu_U, dtype *gpu_V, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_dt,
                              dtype *gpu_alpha, dtype *gpu_gamma, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    //NEED TO DO THIS ONLY FOR FLUID CELLS
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id)
        at(gpu_T, i, j) = (*gpu_dt) * (*gpu_alpha * diffusion(gpu_T_temp, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - convection_Tu(gpu_T_temp, gpu_U, i, j, *gpu_dx, *gpu_dy, *gpu_gamma, gpu_size_x) - convection_Tv(gpu_T_temp, gpu_V, i, j, *gpu_dx, *gpu_dy, *gpu_gamma, gpu_size_x)) + at(gpu_T_temp, i, j);
}

__global__ void calc_fluxes_kernel(dtype *gpu_F, dtype *gpu_G, dtype *gpu_U, dtype *gpu_V, dtype *gpu_T, int *gpu_fluid_id, int *gpu_geometry_data, dtype *gpu_gx, dtype *gpu_gy, dtype *gpu_dx, dtype *gpu_dy, int *gpu_size_x, int *gpu_size_y, dtype *gpu_gamma, dtype *gpu_beta,
                             dtype *gpu_nu, dtype *gpu_dt, bool *gpu_isHeatTransfer) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data,i,j) == *gpu_fluid_id){

        at(gpu_F, i, j) = at(gpu_U, i, j) + (*gpu_dt) * ((*gpu_nu) * diffusion(gpu_U, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - convection_u(gpu_U, gpu_V, *gpu_gamma, i, j, *gpu_dx, *gpu_dy, gpu_size_x) + (*gpu_gx));
        at(gpu_G, i, j) = at(gpu_V, i, j) + (*gpu_dt) * ((*gpu_nu) * diffusion(gpu_V, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - convection_v(gpu_U, gpu_V, *gpu_gamma, i, j, *gpu_dx, *gpu_dy, gpu_size_x) + (*gpu_gy));

        if (*gpu_isHeatTransfer) {

                at(gpu_F,i,j) = at(gpu_F,i,j) - (*gpu_beta) * (*gpu_dt) / 2.0 * (at(gpu_T,i,j) + at(gpu_T,i + 1, j)) * (*gpu_gx) - (*gpu_dt) * (*gpu_gx);
                at(gpu_G,i,j) = at(gpu_G,i,j) - (*gpu_beta) * (*gpu_dt) / 2.0 * (at(gpu_T,i,j) + at(gpu_T,i, j + 1)) * (*gpu_gy) - (*gpu_dt) * (*gpu_gy);
        }
    }
}

__global__ void fluxes_bc_kernel(dtype *gpu_F, dtype *gpu_G, dtype *gpu_U, dtype *gpu_V, int *gpu_fluid_id, int *gpu_fixed_wall_id, int *gpu_adiabatic_id, int *gpu_hot_id, int *gpu_cold_id, int *gpu_moving_wall_id, int *gpu_inflow_id, int *gpu_outflow_id, int *gpu_geometry_data, int *gpu_size_x, int *gpu_size_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < *gpu_size_x && j < *gpu_size_y){

        if (at(gpu_geometry_data, i, j) == *gpu_fixed_wall_id || at(gpu_geometry_data, i, j) == *gpu_adiabatic_id || at(gpu_geometry_data, i, j) == *gpu_hot_id || at(gpu_geometry_data, i, j) == *gpu_cold_id) {
            // B_NE fixed wall corner cell with fluid cells on the North and East directions

            if (check_bound_north(j) && check_bound_east(i) && at(gpu_geometry_data, i, j + 1) == *gpu_fluid_id && at(gpu_geometry_data, i + 1, j) == *gpu_fluid_id) {
                at(gpu_F, i, j) = at(gpu_U, i, j);
                at(gpu_G, i, j) = at(gpu_V, i, j);
            }

            // B_SE fixed wall corner cell with fluid cells on the South and East directions

            else if (check_bound_south(j) && check_bound_east(i) && at(gpu_geometry_data, i, j - 1) == *gpu_fluid_id && at(gpu_geometry_data, i + 1, j) == *gpu_fluid_id) {
                at(gpu_F, i, j) = at(gpu_U, i, j);
                at(gpu_G, i, j - 1) = at(gpu_V, i, j - 1);
            }

            // B_NW fixed wall corner cell with fluid cells on the North and West directions

            else if (check_bound_north(j) && check_bound_west(i) && at(gpu_geometry_data, i, j + 1) == *gpu_fluid_id && at(gpu_geometry_data, i - 1, j) == *gpu_fluid_id) {
                at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);
                at(gpu_G, i, j) = at(gpu_V, i, j);
            }

            // B_SW fixed wall corner cell with fluid cells on the South and West directions

            else if (check_bound_south(j) && check_bound_west(i) && at(gpu_geometry_data, i, j - 1) == *gpu_fluid_id && at(gpu_geometry_data, i - 1, j) == *gpu_fluid_id) {
                at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);
                at(gpu_G, i, j - 1) = at(gpu_V, i, j - 1);
            } 
            else if (check_bound_north(j) && at(gpu_geometry_data, i, j + 1) == *gpu_fluid_id)
                at(gpu_G, i, j) = at(gpu_V, i, j);

            else if (check_bound_south(j) && at(gpu_geometry_data, i, j - 1) == *gpu_fluid_id)
                at(gpu_G,i,j - 1) = at(gpu_V,i,j - 1);

            else if (check_bound_west(i) && at(gpu_geometry_data, i - 1, j) == *gpu_fluid_id)
                at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);

            else if (check_bound_east(i) && at(gpu_geometry_data, i + 1, j) == *gpu_fluid_id)
                at(gpu_F, i, j) = at(gpu_U, i, j);

        }

        else if (at(gpu_geometry_data, i, j) == *gpu_moving_wall_id) {
            at(gpu_G, i, j - 1) = at(gpu_V, i, j - 1);
        } 
        
        else if (at(gpu_geometry_data, i, j) == *gpu_inflow_id) {
            at(gpu_F, i, j) = at(gpu_U, i, j);
        }

        else if (at(gpu_geometry_data, i, j) == *gpu_outflow_id) {

            at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);
        }
    }
}

__global__ void solve_pressure_red_kernel(dtype *gpu_RS, dtype *gpu_P, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_omega, dtype
*gpu_coeff, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    *gpu_coeff = *gpu_omega / (2.0 * (1.0 / (*gpu_dx * *gpu_dx) + 1.0 / (*gpu_dy * *gpu_dy)));

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id && (i + j)%2 != 0){
        at(gpu_P,i, j) = (1.0 - *gpu_omega) * at(gpu_P,i, j) +
                        (*gpu_coeff) * (sor_helper(gpu_P, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - at(gpu_RS,i, j));
    }

}

__global__ void solve_pressure_black_kernel(dtype *gpu_RS, dtype *gpu_P, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_omega, dtype
*gpu_coeff, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    *gpu_coeff = *gpu_omega / (2.0 * (1.0 / (*gpu_dx * *gpu_dx) + 1.0 / (*gpu_dy * *gpu_dy)));

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id && (i + j)%2 == 0){
        at(gpu_P,i, j) = (1.0 - *gpu_omega) * at(gpu_P,i, j) +
                        (*gpu_coeff) * (sor_helper(gpu_P, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - at(gpu_RS,i, j));
    }

}

__global__ void res_kernel(dtype *gpu_RS, dtype *gpu_P, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_res, int *gpu_size_x, int *gpu_size_y, int *gpu_fluid_cells_size, int *gpu_geometry_data){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    dtype val;

    if (i < *gpu_size_x && j < *gpu_size_y){
        at(gpu_res, i, j) = 0.0;
        if (at(gpu_geometry_data, i, j) == *gpu_fluid_id){
            val = laplacian(gpu_P, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - at(gpu_RS,i, j);
            at(gpu_res, i, j) = (val * val);
        }
    }
}

__global__ void calc_rs_kernel(dtype *gpu_RS, dtype *gpu_F, dtype *gpu_G, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_dt, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id)
        at(gpu_RS,i,j) = 1 / (*gpu_dt) * ((at(gpu_F,i, j) - at(gpu_F,i - 1, j)) / (*gpu_dx) + (at(gpu_G,i, j) - at(gpu_G,i, j - 1)) / (*gpu_dy));
}

__global__ void calc_velocities_kernel(dtype *gpu_F, dtype *gpu_G, dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_fluid_id, int *gpu_outflow_id, dtype *gpu_dx, dtype *gpu_dy, int *gpu_size_x, int *gpu_size_y, dtype *gpu_dt, int *gpu_geometry_data)
{   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id)
    {   
        if (( at(gpu_geometry_data, i+1, j)== *gpu_fluid_id) || (at(gpu_geometry_data, i+1, j) == *gpu_outflow_id)) {
            at(gpu_U,i, j) = at(gpu_F,i, j) - ((*gpu_dt)/(*gpu_dx)) * (at(gpu_P,i + 1, j) - at(gpu_P,i, j));           
        }
        if ((at(gpu_geometry_data, i, j+1)==*gpu_fluid_id) || (at(gpu_geometry_data, i, j+1)==*gpu_outflow_id)) {
            at(gpu_V,i, j) = at(gpu_G,i, j) - ((*gpu_dt)/(*gpu_dy)) * (at(gpu_P,i, j + 1) - at(gpu_P,i, j));
        }
    }
}

void CUDA_solver::initialize(Fields &field, Grid &grid, dtype cpu_UIN, dtype cpu_VIN, dtype cpu_wall_temp_a, dtype cpu_wall_temp_h, dtype cpu_wall_temp_c, dtype cpu_omg) {

    UIN = cpu_UIN;
    VIN = cpu_VIN;
    wall_temp_a = cpu_wall_temp_a;
    wall_temp_h = cpu_wall_temp_h;
    wall_temp_c = cpu_wall_temp_c;
    omg = cpu_omg;

    block_size = dim3(BLOCK_SIZE);
    block_size_2d = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    domain_size = (grid.domain().domain_size_x + 2) * (grid.domain().domain_size_y + 2);
    hipMalloc((void **)&gpu_geometry_data, domain_size * sizeof(int));

    grid_size = grid.imaxb() * grid.jmaxb();
    grid_size_x = grid.imaxb();
    grid_size_y = grid.jmaxb();
    grid_fluid_cells_size = grid.fluid_cells().size();

    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);

    hipMalloc((void **)&gpu_T, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_T_temp, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_U, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_V, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_P, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_F, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_G, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_RS, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_res, grid_size * sizeof(dtype));

    hipMalloc((void **)&gpu_dx, sizeof(dtype));
    hipMalloc((void **)&gpu_dy, sizeof(dtype));
    hipMalloc((void **)&gpu_dt, sizeof(dtype));
    hipMalloc((void **)&gpu_alpha, sizeof(dtype));
    hipMalloc((void **)&gpu_gamma, sizeof(dtype));
    hipMalloc((void **)&gpu_beta, sizeof(dtype));
    hipMalloc((void **)&gpu_nu, sizeof(dtype));
    hipMalloc((void **)&gpu_tau, sizeof(dtype));
    hipMalloc((void **)&gpu_gx, sizeof(dtype));
    hipMalloc((void **)&gpu_gy, sizeof(dtype));
    hipMalloc((void **)&gpu_omega, sizeof(dtype));

    hipMalloc((void **)&gpu_coeff, sizeof(dtype));
    hipMalloc((void **)&gpu_rloc, sizeof(dtype));
    hipMalloc((void **)&gpu_val, sizeof(dtype));


    hipMalloc((void **)&gpu_size_x, sizeof(dtype));
    hipMalloc((void **)&gpu_size_y, sizeof(dtype));

    hipMalloc((void **)&gpu_fluid_id, sizeof(int));
    hipMalloc((void **)&gpu_moving_wall_id, sizeof(int));
    hipMalloc((void **)&gpu_fixed_wall_id, sizeof(int));
    hipMalloc((void **)&gpu_inflow_id, sizeof(int));
    hipMalloc((void **)&gpu_outflow_id, sizeof(int));
    hipMalloc((void **)&gpu_adiabatic_id, sizeof(int));
    hipMalloc((void **)&gpu_hot_id, sizeof(int));
    hipMalloc((void **)&gpu_cold_id, sizeof(int));
    hipMalloc((void **)&gpu_fluid_cells_size, sizeof(int));

    hipMalloc((void **)&gpu_wall_temp_a, sizeof(dtype));
    hipMalloc((void **)&gpu_wall_temp_h, sizeof(dtype));
    hipMalloc((void **)&gpu_wall_temp_c, sizeof(dtype));
    hipMalloc((void **)&gpu_isHeatTransfer, sizeof(bool));

    hipMalloc((void **)&gpu_wall_velocity, sizeof(dtype));
    hipMalloc((void **)&gpu_UIN, sizeof(dtype));
    hipMalloc((void **)&gpu_VIN, sizeof(dtype));
    hipMalloc((void **)&gpu_POUT, sizeof(dtype));

    hipMalloc((void **)&gpu_umax, sizeof(dtype));
    hipMalloc((void **)&gpu_vmax, sizeof(dtype));


    // Typecasting raw pointer to thrust device pointer
    thrust_U = thrust::device_pointer_cast(gpu_U);
    thrust_V = thrust::device_pointer_cast(gpu_V);
    thrust_res = thrust::device_pointer_cast(gpu_res);
}

void CUDA_solver::pre_process(Fields &field, Grid &grid, Discretization &discretization, dtype cpu_dt) {

    hipMemcpy(gpu_geometry_data, grid.get_geometry_data().data(), domain_size * sizeof(int), hipMemcpyHostToDevice);

    bool energycheck = field.isHeatTransfer();
    hipMemcpy(gpu_isHeatTransfer, &energycheck, sizeof(bool), hipMemcpyHostToDevice);

    if (energycheck)
        hipMemcpy(gpu_T, field.t_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_U, field.u_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_V, field.v_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_P, field.p_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);

    hipMemcpy(gpu_F, field.f_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_G, field.g_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_RS, field.rs_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);

    dtype var = grid.dx();
    cpu_dx = var;
    hipMemcpy(gpu_dx, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = grid.dy();
    cpu_dy = var;
    hipMemcpy(gpu_dy, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = cpu_dt;
    hipMemcpy(gpu_dt, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_alpha();
    cpu_alpha = var;
    hipMemcpy(gpu_alpha, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_beta();
    hipMemcpy(gpu_beta, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = discretization.get_gamma();
    hipMemcpy(gpu_gamma, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_nu();
    cpu_nu = var;
    hipMemcpy(gpu_nu, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_tau();
    cpu_tau = var;
    hipMemcpy(gpu_tau, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_gx();
    hipMemcpy(gpu_gx, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_gy();
    hipMemcpy(gpu_gy, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = 0;
    hipMemcpy(gpu_umax, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = 0;
    hipMemcpy(gpu_vmax, &var, sizeof(dtype), hipMemcpyHostToDevice);

    hipMemcpy(gpu_size_x, &grid_size_x, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_size_y, &grid_size_y, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_fluid_cells_size, &grid_fluid_cells_size, sizeof(int), hipMemcpyHostToDevice);

    int var1 = GEOMETRY_PGM::moving_wall_id;
    hipMemcpy(gpu_moving_wall_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::fixed_wall_id;
    hipMemcpy(gpu_fixed_wall_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::inflow_id;
    hipMemcpy(gpu_inflow_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::outflow_id;
    hipMemcpy(gpu_outflow_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::hot_id;
    hipMemcpy(gpu_hot_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::cold_id;
    hipMemcpy(gpu_cold_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::adiabatic_id;
    hipMemcpy(gpu_adiabatic_id, &(var1), sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(gpu_wall_temp_a, &wall_temp_a, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_wall_temp_h, &wall_temp_h, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_wall_temp_c, &wall_temp_c, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_omega, &omg, sizeof(dtype), hipMemcpyHostToDevice);

    var = LidDrivenCavity::wall_velocity;
    hipMemcpy(gpu_wall_velocity, &var, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_UIN, &UIN, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_VIN, &VIN, sizeof(dtype), hipMemcpyHostToDevice);
    var = GEOMETRY_PGM::POUT;
    hipMemcpy(gpu_POUT, &var, sizeof(dtype), hipMemcpyHostToDevice);
}

void CUDA_solver::apply_boundary() {


    FixedWallBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_T, gpu_geometry_data, gpu_fluid_id, gpu_moving_wall_id, gpu_fixed_wall_id, gpu_inflow_id, gpu_outflow_id, gpu_adiabatic_id, gpu_hot_id, gpu_cold_id, gpu_wall_temp_a, gpu_wall_temp_h, gpu_wall_temp_c, gpu_isHeatTransfer, gpu_size_x, gpu_size_y);

    MovingWallBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_moving_wall_id, gpu_wall_velocity, gpu_size_x, gpu_size_y, gpu_geometry_data);

    InFlowBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_inflow_id, gpu_UIN, gpu_VIN, gpu_size_x, gpu_size_y, gpu_geometry_data);

    OutFlowBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_outflow_id, gpu_POUT, gpu_size_x, gpu_size_y, gpu_geometry_data);

}

void CUDA_solver::calc_pressure(int max_iter, dtype tolerance, dtype t, dtype dt) {

    dtype res = 0.;
    int iter = 0;

    do{
        apply_boundary();
        solve_pressure_red_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_P, gpu_fluid_id,gpu_dx, gpu_dy, gpu_omega, gpu_coeff, gpu_size_x, gpu_size_y, gpu_geometry_data);
        solve_pressure_black_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_P,gpu_fluid_id, gpu_dx, gpu_dy, gpu_omega, gpu_coeff, gpu_size_x, gpu_size_y, gpu_geometry_data);
        res_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_P, gpu_fluid_id, gpu_dx, gpu_dy, gpu_res, gpu_size_x, gpu_size_y, gpu_fluid_cells_size, gpu_geometry_data);
        res = thrust::reduce(thrust::device, thrust_res, thrust_res + grid_size, (dtype) 0, thrust::plus<dtype>());
        iter++;
    }while(res > tolerance && iter < max_iter);
    
    

    if (iter == max_iter) {
        std::cout << "Max iteration reached at " << t<<" s \n";
    }

    std::cout << "Time = " << std::setw(12) << t << " Residual = "<< std::setw(12) << res <<
    
    " Iter = " << std::setw(8) << iter << " dt = " << std::setw(12) << dt << '\n';

}

void CUDA_solver::calc_T() {
    hipMemcpy(gpu_T_temp, gpu_T, grid_size * sizeof(dtype), hipMemcpyDeviceToDevice);
    calc_T_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_T, gpu_T_temp, gpu_U, gpu_V, gpu_fluid_id, gpu_dx, gpu_dy, gpu_dt, gpu_alpha, gpu_gamma, gpu_size_x, gpu_size_y, gpu_geometry_data);
}

void CUDA_solver::calc_fluxes() {
    calc_fluxes_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_F,gpu_G,gpu_U,gpu_V,gpu_T, gpu_fluid_id, gpu_geometry_data,gpu_gx,gpu_gy,gpu_dx,gpu_dy,gpu_size_x, gpu_size_y, gpu_gamma, gpu_beta, gpu_nu, gpu_dt, gpu_isHeatTransfer);
    fluxes_bc_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_F,gpu_G,gpu_U,gpu_V, gpu_fluid_id, gpu_fixed_wall_id, gpu_adiabatic_id, gpu_hot_id, gpu_cold_id, gpu_moving_wall_id, gpu_inflow_id, gpu_outflow_id, gpu_geometry_data,gpu_size_x, gpu_size_y);
}

void CUDA_solver::calc_rs() {
    calc_rs_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_F,gpu_G, gpu_fluid_id, gpu_dx,gpu_dy, gpu_dt, gpu_size_x, gpu_size_y, gpu_geometry_data);
}

void CUDA_solver::calc_velocities() {
    calc_velocities_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_F, gpu_G, gpu_U, gpu_V, gpu_P, gpu_fluid_id, gpu_outflow_id, gpu_dx, gpu_dy, gpu_size_x, gpu_size_y, gpu_dt, gpu_geometry_data);
}

dtype CUDA_solver::calc_dt() {

    num_blocks = get_num_blocks(grid_size);
    dtype t[4];
    dtype result;

    // Finding max and min element in U and V
    thrust_U_max = thrust::max_element(thrust_U, thrust_U + grid_size);
    thrust_U_min = thrust::min_element(thrust_U, thrust_U + grid_size);
    thrust_V_max = thrust::max_element(thrust_V, thrust_V + grid_size);
    thrust_V_min = thrust::min_element(thrust_V, thrust_V + grid_size);
    
    // Finding Maximum between max element and -(min element) as Abs max needed.
    dtype umax = get_max(*thrust_U_max, -(*thrust_U_min));
    dtype vmax = get_max(*thrust_V_max, -(*thrust_V_min));

    t[0] = 1 / (2 * (cpu_nu) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    t[1] = (cpu_dx) / (umax);
    t[2] = (cpu_dy) / (vmax);   
    t[3] = 1 / (2 * (cpu_alpha) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    dtype temp_dt =t[0];

    // //Previous implementation
    // max_abs_element_kernel<<<num_blocks, block_size>>>(gpu_U, gpu_size_x, gpu_size_y, d_mutex, gpu_umax);
    // max_abs_element_kernel<<<num_blocks, block_size>>>(gpu_V, gpu_size_x, gpu_size_y, d_mutex, gpu_vmax);
    // hipMemcpy((void *)&cpu_umax, gpu_umax, sizeof(dtype), hipMemcpyDeviceToHost);
    // hipMemcpy((void *)&cpu_vmax, gpu_vmax, sizeof(dtype), hipMemcpyDeviceToHost);
    // t[0] = 1 / (2 * (cpu_nu) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    // t[1] = (cpu_dx) / (cpu_umax);
    // t[2] = (cpu_dy) / (cpu_vmax);   
    // t[3] = 1 / (2 * (cpu_alpha) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    // dtype temp_dt =t[0];

    for(int i=1; i<4; i++)
    {
        if(t[i]<temp_dt)
            temp_dt = t[i];
    }

    result = (cpu_tau) * temp_dt;
    hipMemcpy(gpu_dt, &result, sizeof(dtype), hipMemcpyHostToDevice);
    return result;

}

void CUDA_solver::post_process(Fields &field) {
    
    if (field.isHeatTransfer())
        hipMemcpy((void *)field.t_matrix().data(), gpu_T, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    
    hipMemcpy((void *)field.f_matrix().data(), gpu_F, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.g_matrix().data(), gpu_G, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.rs_matrix().data(), gpu_RS, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.u_matrix().data(), gpu_U, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.v_matrix().data(), gpu_V, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.p_matrix().data(), gpu_P, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
}

CUDA_solver::~CUDA_solver() {
    hipFree(gpu_geometry_data);
    hipFree(gpu_T);
    hipFree(gpu_U);
    hipFree(gpu_V);
    hipFree(gpu_P);
    hipFree(gpu_F);
    hipFree(gpu_G);
    hipFree(gpu_RS);
    hipFree(gpu_T_temp);
    hipFree(gpu_dx);
    hipFree(gpu_dy);
    hipFree(gpu_dt);
    hipFree(gpu_gx);
    hipFree(gpu_gy);
    hipFree(gpu_nu);
    hipFree(gpu_beta);
    hipFree(gpu_gamma);
    hipFree(gpu_alpha);
    hipFree(gpu_tau);
    hipFree(gpu_size_x);
    hipFree(gpu_size_y);
    hipFree(gpu_fluid_id);
    hipFree(gpu_fixed_wall_id);
    hipFree(gpu_moving_wall_id);
    hipFree(gpu_inflow_id);
    hipFree(gpu_outflow_id);
    hipFree(gpu_adiabatic_id);
    hipFree(gpu_hot_id);
    hipFree(gpu_cold_id);
    hipFree(gpu_wall_temp_a);
    hipFree(gpu_wall_temp_c);
    hipFree(gpu_wall_temp_h);
    hipFree(gpu_isHeatTransfer);
    hipFree(gpu_UIN);
    hipFree(gpu_VIN);
    hipFree(gpu_POUT);
    hipFree(gpu_wall_velocity);
    hipFree(gpu_umax);
    hipFree(gpu_vmax);
    hipFree(gpu_omega);
    hipFree(gpu_coeff);
    hipFree(gpu_rloc);
    hipFree(gpu_val);
    hipFree(gpu_res);
    hipFree(gpu_fluid_cells_size);
}
