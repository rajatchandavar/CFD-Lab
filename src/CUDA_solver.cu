#include "hip/hip_runtime.h"
#include "CUDA_solver.cuh"
#include <cmath>
#include <iostream>
#include <iomanip> 
#include <math.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif
#define at(var, i, j) var[(j) * (*gpu_size_x) + (i)]

#define check_bound_north(j) ((j + 1) < *gpu_size_y)
#define check_bound_south(j) ((j - 1) >= 0)
#define check_bound_east(i) ((i + 1) < *gpu_size_x)
#define check_bound_west(i) ((i - 1) >=0)

__device__ dtype interpolate(dtype *A, int i, int j, int i_offset, int j_offset, int *gpu_size_x) {
    dtype result = (at(A, i, j) + at(A, i + i_offset, j + j_offset)) / 2;
    return result;
}

__device__ dtype diffusion(dtype *A, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {
    dtype result = (at(A, i + 1, j) - 2.0 * at(A, i, j) + at(A, i - 1, j)) / (gpu_dx * gpu_dx) +
                    (at(A, i, j + 1) - 2.0 * at(A, i, j) + at(A, i, j - 1)) / (gpu_dy * gpu_dy);

    return result;
}

//NO NEED TO PASS ALL OF gpu_U, V only surrounding of i,j sufficient
__device__ dtype convection_u(dtype *gpu_U, dtype *gpu_V, dtype gpu_gamma, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {

    dtype t1 = interpolate(gpu_U, i, j, 1, 0, gpu_size_x);
    dtype t2 = interpolate(gpu_U, i, j, -1, 0, gpu_size_x);
    dtype du2_dx = 1 / gpu_dx * ((t1 * t1) - (t2 * t2)) + gpu_gamma / gpu_dx *\
            ((fabsf(interpolate(gpu_U, i, j, 1, 0, gpu_size_x)) * (at(gpu_U, i, j) - at(gpu_U, i + 1, j)) / 2) -\
             fabsf(interpolate(gpu_U, i, j, -1, 0, gpu_size_x)) * (at(gpu_U, i - 1, j) - at(gpu_U, i, j)) / 2);
    dtype duv_dy = 1 / gpu_dy * (((interpolate(gpu_V, i, j, 1, 0, gpu_size_x)) * (interpolate(gpu_U, i, j, 0, 1, gpu_size_x))) -\
                         ((interpolate(gpu_V, i, j - 1, 1, 0, gpu_size_x)) * (interpolate(gpu_U, i, j, 0, -1, gpu_size_x)))) +\
                    gpu_gamma / gpu_dy *\
                        ((fabsf(interpolate(gpu_V, i, j, 1, 0, gpu_size_x)) * (at(gpu_U, i, j) - at(gpu_U, i, j + 1)) / 2) -\
                         (fabsf(interpolate(gpu_V, i, j - 1, 1, 0, gpu_size_x)) * (at(gpu_U, i, j - 1) - at(gpu_U, i, j)) / 2));

    dtype result = du2_dx + duv_dy;
    return result;
}

__device__ dtype convection_v(dtype *gpu_U, dtype *gpu_V, dtype gpu_gamma, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {

    dtype t1 = interpolate(gpu_V, i, j, 0, 1, gpu_size_x);
    dtype t2 = interpolate(gpu_V, i, j - 1, 0, 1, gpu_size_x);
    dtype dv2_dy = 1 / gpu_dy * ((t1 * t1) - (t2 * t2)) + gpu_gamma / gpu_dy *\
            ((fabsf(interpolate(gpu_V, i, j, 0, 1, gpu_size_x)) * (at(gpu_V, i, j) - at(gpu_V, i, j + 1)) / 2) -\
             fabsf(interpolate(gpu_V, i, j - 1, 0, 1, gpu_size_x)) * (at(gpu_V, i, j - 1) - at(gpu_V, i, j)) / 2);
    dtype duv_dx = 1 / gpu_dx *\
                        (((interpolate(gpu_U, i, j, 0, 1, gpu_size_x)) * (interpolate(gpu_V, i, j, 1, 0, gpu_size_x))) -\
                         ((interpolate(gpu_U, i - 1, j, 0, 1, gpu_size_x)) * (interpolate(gpu_V, i - 1, j, 1, 0, gpu_size_x)))) +\
                    gpu_gamma / gpu_dx *\
                        ((fabsf(interpolate(gpu_U, i, j, 0, 1, gpu_size_x)) * (at(gpu_V, i, j) - at(gpu_V, i + 1, j)) / 2) -\
                         (fabsf(interpolate(gpu_U, i - 1, j, 0, 1, gpu_size_x)) * (at(gpu_V, i - 1, j) - at(gpu_V, i, j)) / 2));

    dtype result = dv2_dy + duv_dx;
    return result;
}

__device__ dtype convection_Tu(dtype *gpu_T, dtype *gpu_U, int i, int j, dtype gpu_dx, dtype gpu_dy, dtype gpu_gamma, int *gpu_size_x) {
    dtype result;
    result = 1 / gpu_dx * (at(gpu_U, i, j) * interpolate(gpu_T, i, j, 1, 0, gpu_size_x) - at(gpu_U, i - 1, j) * interpolate(gpu_T, i - 1, j, 1, 0, gpu_size_x)) +
             gpu_gamma / gpu_dx * (fabsf(at(gpu_U, i, j)) * (at(gpu_T, i, j) - at(gpu_T, i + 1, j)) / 2 - fabsf(at(gpu_U, i - 1, j)) * (at(gpu_T, i - 1, j) - at(gpu_T, i, j)) / 2);
    return result;
}

__device__ dtype convection_Tv(dtype *gpu_T, dtype *gpu_V, int i, int j, dtype gpu_dx, dtype gpu_dy, dtype gpu_gamma, int *gpu_size_x) {
    dtype result;
    result = 1 / gpu_dy * (at(gpu_V, i, j) * interpolate(gpu_T, i, j, 0, 1, gpu_size_x) - at(gpu_V, i, j - 1) * interpolate(gpu_T, i, j - 1, 0, 1, gpu_size_x)) +
             gpu_gamma / gpu_dy * (fabsf(at(gpu_V, i, j)) * (at(gpu_T, i, j) - at(gpu_T, i, j + 1)) / 2 - fabsf(at(gpu_V, i, j - 1)) * (at(gpu_T, i, j - 1) - at(gpu_T, i, j)) / 2);
    return result;
}

__device__ dtype sor_helper(dtype *gpu_P, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {
    dtype result = (at(gpu_P,i + 1, j) + at(gpu_P,i - 1, j)) / (gpu_dx * gpu_dx) + (at(gpu_P,i, j + 1) + at(gpu_P,i, j - 1)) / (gpu_dy * gpu_dy);
    return result;
}

__device__ dtype laplacian(dtype *gpu_P, int i, int j, dtype gpu_dx, dtype gpu_dy, int *gpu_size_x) {
    dtype result = (at(gpu_P,i + 1, j) - 2.0 * at(gpu_P,i, j) + at(gpu_P,i - 1, j)) / (gpu_dx * gpu_dx) +
                   (at(gpu_P,i, j + 1) - 2.0 * at(gpu_P,i, j) + at(gpu_P,i, j - 1)) / (gpu_dy * gpu_dy);
    return result;
}

dim3 CUDA_solver::get_num_blocks(int size) { return (size + BLOCK_SIZE - 1) / BLOCK_SIZE; }

dim3 CUDA_solver::get_num_blocks_2d(int gpu_size_x, int gpu_size_y) {
    return (dim3((gpu_size_x + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X, (gpu_size_y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y));
}

__global__ void FixedWallBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, dtype *gpu_T, int *gpu_geometry_data,
int *gpu_fluid_id, int *gpu_moving_wall_id, int *gpu_fixed_wall_id, int *gpu_inflow_id, int *gpu_outflow_id, int *gpu_adiabatic_id, int *gpu_hot_id,
int *gpu_cold_id, dtype *gpu_wall_temp_a, dtype *gpu_wall_temp_h, dtype *gpu_wall_temp_c, bool *gpu_isHeatTransfer, int *gpu_size_x, int
*gpu_size_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && (at(gpu_geometry_data, i, j) == 3 || at(gpu_geometry_data, i, j) == 5 || at(gpu_geometry_data, i, j) == 6 || at(gpu_geometry_data, i, j) == 7)) {
        // obstacles B_NE (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the North and East directions 

        if(check_bound_north(j) && check_bound_east(i) && at(gpu_geometry_data,i,j+1)==*gpu_fluid_id && at(gpu_geometry_data,i+1,j)==*gpu_fluid_id) {
            at(gpu_U, i, j) = 0.0;
            at(gpu_U, i - 1, j) = -at(gpu_U, i - 1, j + 1);
            at(gpu_V, i, j) = 0.0;
            at(gpu_V, i, j - 1) = -at(gpu_V, i + 1, j - 1);
            at(gpu_P, i, j) = (at(gpu_P, i, j + 1) + at(gpu_P, i + 1, j))/2;

            if(*gpu_isHeatTransfer==1) {
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T, i, j) = (at(gpu_T, i + 1, j) + at(gpu_T, i, j + 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j + 1) + at(gpu_T,i + 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j + 1) + at(gpu_T,i + 1, j) )/2;
            }
        }

        // obstacles B_SE (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the South and East directions 

        else if(check_bound_south(j) && check_bound_east(i) && at(gpu_geometry_data,i,j-1)==*gpu_fluid_id&& at(gpu_geometry_data,i+1,j)==*gpu_fluid_id) {

            at(gpu_U, i, j) = 0.0;
            at(gpu_U, i - 1, j) = -at(gpu_U, i - 1, j - 1);
            at(gpu_V, i, j - 1) = 0.0;
            at(gpu_V, i, j) = -at(gpu_V, i + 1, j);
            at(gpu_P,i, j) = (at(gpu_P,i + 1, j) + at(gpu_P,i, j - 1))/2;

            if(*gpu_isHeatTransfer==1) {
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = (at(gpu_T, i + 1, j) + at(gpu_T, i, j - 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T, i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j - 1) + at(gpu_T,i + 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j - 1) + at(gpu_T,i + 1, j) )/2;
            }

        }

        // obstacle B_NW (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the North and West directions 
            
        else if(check_bound_north(j) && check_bound_west(i) && at(gpu_geometry_data,i,j+1)==*gpu_fluid_id && at(gpu_geometry_data,i-1,j)==*gpu_fluid_id) {

            at(gpu_U,i - 1, j) = 0.0;
            at(gpu_U,i, j) = -at(gpu_U,i, j + 1);
            at(gpu_V,i, j) = 0.0;
            at(gpu_V,i, j - 1) = -at(gpu_V,i - 1, j - 1);
            at(gpu_P,i,j) = (at(gpu_P,i - 1, j) + at(gpu_P,i, j + 1))/2;

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = (at(gpu_T,i - 1, j) + at(gpu_T,i, j + 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j + 1) + at(gpu_T,i - 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j + 1) + at(gpu_T,i - 1, j) )/2;
            }

        }

        // obstacle B_SW (Corner Cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the South and West directions 

        else if(check_bound_south(j) && check_bound_west(i) && at(gpu_geometry_data,i,j-1)==*gpu_fluid_id&& at(gpu_geometry_data,i-1,j)==*gpu_fluid_id){
            at(gpu_U,i - 1, j) = 0.0;
            at(gpu_U,i, j) = at(gpu_U,i, j - 1);
            at(gpu_V,i, j - 1) = 0.0;
            at(gpu_V,i, j) = -at(gpu_V,i - 1, j);
            at(gpu_P,i, j) = (at(gpu_P,i - 1, j) + at(gpu_P,i, j - 1))/2;
           
            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                   at(gpu_T,i, j) = (at(gpu_T,i - 1, j) + at(gpu_T,i, j - 1))/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - (at(gpu_T,i, j - 1) + at(gpu_T,i - 1, j) )/2;
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - (at(gpu_T,i, j - 1) + at(gpu_T,i - 1, j) )/2;
            }
            
        }

        // Bottom Wall B_N (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the North direction

        else if(check_bound_north(j) && at(gpu_geometry_data,i,j+1)==*gpu_fluid_id){
            at(gpu_U,i, j) = -at(gpu_U,i, j + 1);
            at(gpu_V,i, j) = 0.0;
            at(gpu_P,i, j) = at(gpu_P,i, j + 1);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i, j + 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - at(gpu_T,i, j + 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - at(gpu_T,i, j + 1);
            }
        }

        // Top Wall B_S (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the South direction

        else if(check_bound_south(j) && at(gpu_geometry_data,i,j-1)==*gpu_fluid_id){

            at(gpu_U,i, j) = -at(gpu_U,i, j - 1);
            at(gpu_V,i, j) = 0.0;
            at(gpu_P,i, j) = at(gpu_P,i, j - 1);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i, j - 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2 * (*gpu_wall_temp_h) - at(gpu_T,i, j - 1);
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2 * (*gpu_wall_temp_c) - at(gpu_T,i, j - 1);
            }
        }

        // Left Wall B_E (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on 
        // the East direction

        else if(check_bound_east(i) && at(gpu_geometry_data,i+1,j)==*gpu_fluid_id){
            at(gpu_U,i, j) = 0.0;
            at(gpu_V,i, j) = -at(gpu_V,i + 1, j);
            at(gpu_P,i, j) = at(gpu_P,i + 1, j);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i + 1, j);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id) 
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - at(gpu_T,i + 1, j);        
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - at(gpu_T,i + 1, j);
            }
        }

        
        /***********************************************************************************************
        * Right Wall B_W (edge cell) - This section applies the appropriate boundary conditions to a fixed wall with fluid cells on the West direction *
        ***********************************************************************************************/

        else if(check_bound_west(i) && at(gpu_geometry_data,i-1,j)==0){
            //Since u grid is staggered, the u velocity of cells to left of ghost layer should be set to 0.
            at(gpu_U,i - 1, j) = 0.0; 
            at(gpu_V,i, j) = -at(gpu_V,i - 1, j);
            at(gpu_P,i, j) = at(gpu_P,i - 1, j);

            if(*gpu_isHeatTransfer==1){
                if(at(gpu_geometry_data,i,j) == *gpu_adiabatic_id)
                    at(gpu_T,i, j) = at(gpu_T,i - 1, j);
                else if (at(gpu_geometry_data,i,j) == *gpu_hot_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_h) - at(gpu_T,i - 1, j);
                else if (at(gpu_geometry_data,i,j) == *gpu_cold_id)
                    at(gpu_T,i, j) = 2*(*gpu_wall_temp_c) - at(gpu_T,i - 1, j);
            }
        }
    }
}

__global__ void MovingWallBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_moving_wall_id, dtype *gpu_wall_velocity, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_moving_wall_id && check_bound_south(j)) {       
        at(gpu_U,i, j) = 2*(*gpu_wall_velocity)- at(gpu_U,i, j-1);
        //Since v grid is staggered, the v velocity of cells to below of ghost layer should be set to 0.
        at(gpu_V,i,j - 1) = 0.0;
        at(gpu_P,i,j) = at(gpu_P,i, j-1);
    }
}

__global__ void InFlowBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_inflow_id, dtype *gpu_UIN, dtype *gpu_VIN, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (check_bound_east(i) && i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_inflow_id) {
        at(gpu_U,i,j) = (*gpu_UIN);
        at(gpu_V,i,j) = 2*(*gpu_VIN) - at(gpu_V,i + 1, j);
        at(gpu_P,i,j) = at(gpu_P,i + 1, j);
    }
}

__global__ void OutFlowBoundary(dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_outflow_id, dtype *gpu_POUT, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (check_bound_west(i) && i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_outflow_id) {
            at(gpu_U,i,j) = at(gpu_U,i - 1,j);
            at(gpu_V,i,j) = at(gpu_V,i - 1,j);
            at(gpu_P,i,j) = 2*(*gpu_POUT) - at(gpu_P,i - 1, j);
    }
}

__global__ void calc_T_kernel(dtype *gpu_T, dtype *gpu_T_temp, dtype *gpu_U, dtype *gpu_V, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_dt,
                              dtype *gpu_alpha, dtype *gpu_gamma, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    //NEED TO DO THIS ONLY FOR FLUID CELLS
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id)
        at(gpu_T, i, j) = (*gpu_dt) * (*gpu_alpha * diffusion(gpu_T_temp, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - convection_Tu(gpu_T_temp, gpu_U, i, j, *gpu_dx, *gpu_dy, *gpu_gamma, gpu_size_x) - convection_Tv(gpu_T_temp, gpu_V, i, j, *gpu_dx, *gpu_dy, *gpu_gamma, gpu_size_x)) + at(gpu_T_temp, i, j);
}

__global__ void calc_fluxes_kernel(dtype *gpu_F, dtype *gpu_G, dtype *gpu_U, dtype *gpu_V, dtype *gpu_T, int *gpu_fluid_id, int *gpu_geometry_data, dtype *gpu_gx, dtype *gpu_gy, dtype *gpu_dx, dtype *gpu_dy, int *gpu_size_x, int *gpu_size_y, dtype *gpu_gamma, dtype *gpu_beta,
                             dtype *gpu_nu, dtype *gpu_dt, bool *gpu_isHeatTransfer) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data,i,j) == *gpu_fluid_id){

        at(gpu_F, i, j) = at(gpu_U, i, j) + (*gpu_dt) * ((*gpu_nu) * diffusion(gpu_U, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - convection_u(gpu_U, gpu_V, *gpu_gamma, i, j, *gpu_dx, *gpu_dy, gpu_size_x) + (*gpu_gx));
        at(gpu_G, i, j) = at(gpu_V, i, j) + (*gpu_dt) * ((*gpu_nu) * diffusion(gpu_V, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - convection_v(gpu_U, gpu_V, *gpu_gamma, i, j, *gpu_dx, *gpu_dy, gpu_size_x) + (*gpu_gy));

        if (*gpu_isHeatTransfer) {

                at(gpu_F,i,j) = at(gpu_F,i,j) - (*gpu_beta) * (*gpu_dt) / 2.0 * (at(gpu_T,i,j) + at(gpu_T,i + 1, j)) * (*gpu_gx) - (*gpu_dt) * (*gpu_gx);
                at(gpu_G,i,j) = at(gpu_G,i,j) - (*gpu_beta) * (*gpu_dt) / 2.0 * (at(gpu_T,i,j) + at(gpu_T,i, j + 1)) * (*gpu_gy) - (*gpu_dt) * (*gpu_gy);
        }
    }
}

__global__ void fluxes_bc_kernel(dtype *gpu_F, dtype *gpu_G, dtype *gpu_U, dtype *gpu_V, int *gpu_fluid_id, int *gpu_fixed_wall_id, int *gpu_adiabatic_id, int *gpu_hot_id, int *gpu_cold_id, int *gpu_moving_wall_id, int *gpu_inflow_id, int *gpu_outflow_id, int *gpu_geometry_data, int *gpu_size_x, int *gpu_size_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < *gpu_size_x && j < *gpu_size_y){

        if (at(gpu_geometry_data, i, j) == *gpu_fixed_wall_id || at(gpu_geometry_data, i, j) == *gpu_adiabatic_id || at(gpu_geometry_data, i, j) == *gpu_hot_id || at(gpu_geometry_data, i, j) == *gpu_cold_id) {
            // B_NE fixed wall corner cell with fluid cells on the North and East directions

            if (check_bound_north(j) && check_bound_east(i) && at(gpu_geometry_data, i, j + 1) == *gpu_fluid_id && at(gpu_geometry_data, i + 1, j) == *gpu_fluid_id) {
                at(gpu_F, i, j) = at(gpu_U, i, j);
                at(gpu_G, i, j) = at(gpu_V, i, j);
            }

            // B_SE fixed wall corner cell with fluid cells on the South and East directions

            else if (check_bound_south(j) && check_bound_east(i) && at(gpu_geometry_data, i, j - 1) == *gpu_fluid_id && at(gpu_geometry_data, i + 1, j) == *gpu_fluid_id) {
                at(gpu_F, i, j) = at(gpu_U, i, j);
                at(gpu_G, i, j - 1) = at(gpu_V, i, j - 1);
            }

            // B_NW fixed wall corner cell with fluid cells on the North and West directions

            else if (check_bound_north(j) && check_bound_west(i) && at(gpu_geometry_data, i, j + 1) == *gpu_fluid_id && at(gpu_geometry_data, i - 1, j) == *gpu_fluid_id) {
                at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);
                at(gpu_G, i, j) = at(gpu_V, i, j);
            }

            // B_SW fixed wall corner cell with fluid cells on the South and West directions

            else if (check_bound_south(j) && check_bound_west(i) && at(gpu_geometry_data, i, j - 1) == *gpu_fluid_id && at(gpu_geometry_data, i - 1, j) == *gpu_fluid_id) {
                at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);
                at(gpu_G, i, j - 1) = at(gpu_V, i, j - 1);
            } 
            else if (check_bound_north(j) && at(gpu_geometry_data, i, j + 1) == *gpu_fluid_id)
                at(gpu_G, i, j) = at(gpu_V, i, j);

            else if (check_bound_south(j) && at(gpu_geometry_data, i, j - 1) == *gpu_fluid_id)
                at(gpu_G,i,j - 1) = at(gpu_V,i,j - 1);

            else if (check_bound_west(i) && at(gpu_geometry_data, i - 1, j) == *gpu_fluid_id)
                at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);

            else if (check_bound_east(i) && at(gpu_geometry_data, i + 1, j) == *gpu_fluid_id)
                at(gpu_F, i, j) = at(gpu_U, i, j);

        }

        else if (at(gpu_geometry_data, i, j) == *gpu_moving_wall_id) {
            at(gpu_G, i, j - 1) = at(gpu_V, i, j - 1);
        } 
        
        else if (at(gpu_geometry_data, i, j) == *gpu_inflow_id) {
            at(gpu_F, i, j) = at(gpu_U, i, j);
        }

        else if (at(gpu_geometry_data, i, j) == *gpu_outflow_id) {

            at(gpu_F, i - 1, j) = at(gpu_U, i - 1, j);
        }
    }
}

__global__ void solve_pressure_red_kernel(dtype *gpu_RS, dtype *gpu_P, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_omega, dtype
*gpu_coeff, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    *gpu_coeff = *gpu_omega / (2.0 * (1.0 / (*gpu_dx * *gpu_dx) + 1.0 / (*gpu_dy * *gpu_dy)));

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id && (i + j)%2 != 0){
        at(gpu_P,i, j) = (1.0 - *gpu_omega) * at(gpu_P,i, j) +
                        (*gpu_coeff) * (sor_helper(gpu_P, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - at(gpu_RS,i, j));
    }

}

__global__ void solve_pressure_black_kernel(dtype *gpu_RS, dtype *gpu_P, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_omega, dtype
*gpu_coeff, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    *gpu_coeff = *gpu_omega / (2.0 * (1.0 / (*gpu_dx * *gpu_dx) + 1.0 / (*gpu_dy * *gpu_dy)));

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id && (i + j)%2 == 0){
        at(gpu_P,i, j) = (1.0 - *gpu_omega) * at(gpu_P,i, j) +
                        (*gpu_coeff) * (sor_helper(gpu_P, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - at(gpu_RS,i, j));
    }

}

__global__ void calc_res_kernel(dtype *gpu_RS, dtype *gpu_P, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_res, int *gpu_size_x, int *gpu_size_y, int *gpu_fluid_cells_size, int *gpu_geometry_data){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    dtype val, rloc;

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id){
        val = laplacian(gpu_P, i, j, *gpu_dx, *gpu_dy, gpu_size_x) - at(gpu_RS,i, j);
        rloc = (val * val);
    }


    //Reduction operation of Residuals

    //Local to each block
    __shared__ dtype resdata[BLOCK_SIZE_X][BLOCK_SIZE_Y];

    //Filling resdata in a block
    if (i == 0 && j == 0) {
        *gpu_res = 0;
    }
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id) {
        resdata[tid_x][tid_y] = rloc;
    } 
    else {
        resdata[tid_x][tid_y] = 0.;
    }
    __syncthreads();

    // Reduction by first reducing along x dimesion and then y dimension. 
    // Finally resdata[0][0] in each block will have sum of that block
    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (tid_x < s) {
            resdata[tid_x][tid_y] = resdata[tid_x][tid_y] + resdata[tid_x + s][tid_y];
        }
        __syncthreads();
    }

    for (int r = blockDim.y / 2; r > 0; r /= 2) {
        if (tid_y < r) {
            resdata[0][tid_y] = resdata[0][tid_y] + resdata[0][tid_y + r];
        }
        __syncthreads();
    }

    //Adding resdata[0][0] among all blocks
    if (tid_x == 0 && tid_y == 0) {
        atomicAdd(gpu_res, resdata[0][0]);
        *gpu_res = *gpu_res / *gpu_fluid_cells_size;
        *gpu_res = sqrt(*gpu_res);
    }
}

__global__ void calc_rs_kernel(dtype *gpu_RS, dtype *gpu_F, dtype *gpu_G, int *gpu_fluid_id, dtype *gpu_dx, dtype *gpu_dy, dtype *gpu_dt, int *gpu_size_x, int *gpu_size_y, int *gpu_geometry_data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id)
        at(gpu_RS,i,j) = 1 / (*gpu_dt) * ((at(gpu_F,i, j) - at(gpu_F,i - 1, j)) / (*gpu_dx) + (at(gpu_G,i, j) - at(gpu_G,i, j - 1)) / (*gpu_dy));
}

__global__ void calc_velocities_kernel(dtype *gpu_F, dtype *gpu_G, dtype *gpu_U, dtype *gpu_V, dtype *gpu_P, int *gpu_fluid_id, int *gpu_outflow_id, dtype *gpu_dx, dtype *gpu_dy, int *gpu_size_x, int *gpu_size_y, dtype *gpu_dt, int *gpu_geometry_data)
{   
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < *gpu_size_x && j < *gpu_size_y && at(gpu_geometry_data, i, j) == *gpu_fluid_id)
    {   
        if (( at(gpu_geometry_data, i+1, j)== *gpu_fluid_id) || (at(gpu_geometry_data, i+1, j) == *gpu_outflow_id)) {
            at(gpu_U,i, j) = at(gpu_F,i, j) - ((*gpu_dt)/(*gpu_dx)) * (at(gpu_P,i + 1, j) - at(gpu_P,i, j));           
        }
        if ((at(gpu_geometry_data, i, j+1)==*gpu_fluid_id) || (at(gpu_geometry_data, i, j+1)==*gpu_outflow_id)) {
            at(gpu_V,i, j) = at(gpu_G,i, j) - ((*gpu_dt)/(*gpu_dy)) * (at(gpu_P,i, j + 1) - at(gpu_P,i, j));
        }
    }
}

__global__ void max_abs_element_kernel(dtype *array, int *gpu_size_x, int *gpu_size_y, int *d_mutex, dtype *array_max)
{

    unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	__shared__ dtype cache[BLOCK_SIZE]; //block size
	dtype temp = 0;
	while(index + offset < (*gpu_size_x)*(*gpu_size_y)){
		temp = fmaxf(temp, fabsf(array[index + offset]));

		offset += stride;
	}

	cache[threadIdx.x] = temp;

	__syncthreads();


    unsigned int i = blockDim.x/2;
	while(i != 0){
		if(threadIdx.x < i){
			cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
		}

		__syncthreads();
		i /= 2;
	}

	if(threadIdx.x == 0){
		while(atomicCAS(d_mutex,0,1) != 0);  //lock
		*array_max = fmaxf(*array_max, cache[0]);
		atomicExch(d_mutex, 0);  //unlock
	}
}

__global__ void reset_pressure(dtype *gpu_P, dtype *gpu_qr_P, int *gpu_fluid_id, int *gpu_geometry_data, int *gpu_size_x, int *gpu_size_y) {

   int k = 0;
   
   for (int j = 0; j < *gpu_size_y; j++) {
     for (int i = 0; i < *gpu_size_x; i++) {
      if ( at(gpu_geometry_data, i, j) == *gpu_fluid_id ) {
        at(gpu_P, i, j) = gpu_qr_P[k];
        k++;
       }
    }
}
}

__global__ void update_RS(dtype *gpu_RS, dtype *gpu_qr_RS, dtype *gpu_P, int *gpu_size_x, int *gpu_size_y, int
*gpu_geometry_data, dtype *gpu_dx, dtype *gpu_dy, int *gpu_fluid_id) {
 
    int k = 0;

    for (int j = 0; j < *gpu_size_y; j++) {
        for (int i = 0; i < *gpu_size_x; i++) {
            if( at(gpu_geometry_data, i, j) == *gpu_fluid_id) {
                gpu_qr_RS[k] = at(gpu_RS, i, j); // B
                if(j-1 == 0) {
                    gpu_qr_RS[k] -= at(gpu_P,i,j-1)/(*gpu_dy * (*gpu_dy)); //B
                }

                if(i-1 == 0) {
                        gpu_qr_RS[k] -= at(gpu_P,i-1,j)/(*gpu_dx * (*gpu_dx)); // L
    
                }

                if (i==*gpu_size_x-2) {
                        gpu_qr_RS[k] -= at(gpu_P,i+1,j)/(*gpu_dx * (*gpu_dx)); // R

                }

                if(j == *gpu_size_y-2) {
                        gpu_qr_RS[k] -= at(gpu_P,i,j+1)/(*gpu_dy * (*gpu_dy)); //T
                }

                k++;

            }
        }
    }
}

void CUDA_solver::solve_pressure_cusolver(dtype *gpu_csrValA, int *gpu_csrRowPtrA,  int *gpu_csrColIndA, dtype *gpu_qr_RS, dtype *gpu_qr_P, int n, int nnzA) {
    hipsolverSpHandle_t handleSolver;
    hipsolverStatus_t Checker = hipsolverSpCreate(&handleSolver);
    
    hipsparseMatDescr_t descrA = 0;
    descrA = 0;
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatDiagType(descrA, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    int reorder = 0;
    dtype tol = 1e-6;
    
    int valuefor,*singularity = &valuefor;
    *singularity = 0;
    
    hipStream_t streamId = NULL;
    hipsolverStatus_t cudasu = hipsolverSpSetStream(handleSolver, streamId);

    hipsolverStatus_t result = hipsolverSpScsrlsvqr(handleSolver, n, nnzA, descrA, gpu_csrValA, gpu_csrRowPtrA,
    gpu_csrColIndA, gpu_qr_RS, tol, reorder, gpu_qr_P, singularity); 

    hipsolverStatus_t hipsolverSpDestroy(hipsolverSpHandle_t handleSolver);

}

void CUDA_solver::initialize(Fields &field, Grid &grid, dtype cpu_UIN, dtype cpu_VIN, dtype cpu_wall_temp_a, dtype cpu_wall_temp_h, dtype cpu_wall_temp_c, dtype cpu_omg) {

    UIN = cpu_UIN;
    VIN = cpu_VIN;
    wall_temp_a = cpu_wall_temp_a;
    wall_temp_h = cpu_wall_temp_h;
    wall_temp_c = cpu_wall_temp_c;
    omg = cpu_omg;
    nnzA = field.get_nnzA();

    block_size = dim3(BLOCK_SIZE);
    block_size_2d = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y);

    domain_size = (grid.domain().domain_size_x + 2) * (grid.domain().domain_size_y + 2);
    hipMalloc((void **)&gpu_geometry_data, domain_size * sizeof(int));

    grid_size = grid.imaxb() * grid.jmaxb();
    grid_size_x = grid.imaxb();
    grid_size_y = grid.jmaxb();
    grid_fluid_cells_size = grid.fluid_cells().size();
    n = grid_fluid_cells_size;

    hipMalloc((void **)&gpu_T, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_T_temp, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_U, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_V, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_P, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_F, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_G, grid_size * sizeof(dtype));
    hipMalloc((void **)&gpu_RS, grid_size * sizeof(dtype));

    hipMalloc((void**)&gpu_qr_RS, grid_fluid_cells_size * sizeof(dtype));
    hipMalloc((void**)&gpu_qr_P, grid_fluid_cells_size * sizeof(dtype));

    hipMalloc((void **)&gpu_csrValA, nnzA * sizeof(dtype));
    hipMalloc((void **)&gpu_csrColIndA, nnzA * sizeof(int));
    hipMalloc((void **)&gpu_csrRowPtrA, (n + 1) * sizeof(int));

    hipMalloc((void **)&gpu_nnzA, sizeof(int));
    hipMalloc((void **)&gpu_n, sizeof(int));

    hipMalloc((void **)&gpu_dx, sizeof(dtype));
    hipMalloc((void **)&gpu_dy, sizeof(dtype));
    hipMalloc((void **)&gpu_dt, sizeof(dtype));
    hipMalloc((void **)&gpu_alpha, sizeof(dtype));
    hipMalloc((void **)&gpu_gamma, sizeof(dtype));
    hipMalloc((void **)&gpu_beta, sizeof(dtype));
    hipMalloc((void **)&gpu_nu, sizeof(dtype));
    hipMalloc((void **)&gpu_tau, sizeof(dtype));
    hipMalloc((void **)&gpu_gx, sizeof(dtype));
    hipMalloc((void **)&gpu_gy, sizeof(dtype));
    hipMalloc((void **)&gpu_omega, sizeof(dtype));

    hipMalloc((void **)&gpu_coeff, sizeof(dtype));
    hipMalloc((void **)&gpu_rloc, sizeof(dtype));
    hipMalloc((void **)&gpu_val, sizeof(dtype));
    hipMalloc((void **)&gpu_res, sizeof(dtype));

    hipMalloc((void **)&gpu_size_x, sizeof(dtype));
    hipMalloc((void **)&gpu_size_y, sizeof(dtype));

    hipMalloc((void **)&gpu_fluid_id, sizeof(int));
    hipMalloc((void **)&gpu_moving_wall_id, sizeof(int));
    hipMalloc((void **)&gpu_fixed_wall_id, sizeof(int));
    hipMalloc((void **)&gpu_inflow_id, sizeof(int));
    hipMalloc((void **)&gpu_outflow_id, sizeof(int));
    hipMalloc((void **)&gpu_adiabatic_id, sizeof(int));
    hipMalloc((void **)&gpu_hot_id, sizeof(int));
    hipMalloc((void **)&gpu_cold_id, sizeof(int));
    hipMalloc((void **)&gpu_fluid_cells_size, sizeof(int));

    hipMalloc((void **)&gpu_wall_temp_a, sizeof(dtype));
    hipMalloc((void **)&gpu_wall_temp_h, sizeof(dtype));
    hipMalloc((void **)&gpu_wall_temp_c, sizeof(dtype));
    hipMalloc((void **)&gpu_isHeatTransfer, sizeof(bool));
    hipMalloc((void**)&d_mutex, sizeof(int));

    hipMalloc((void **)&gpu_wall_velocity, sizeof(dtype));
    hipMalloc((void **)&gpu_UIN, sizeof(dtype));
    hipMalloc((void **)&gpu_VIN, sizeof(dtype));
    hipMalloc((void **)&gpu_POUT, sizeof(dtype));

    hipMalloc((void **)&gpu_umax, sizeof(dtype));
    hipMalloc((void **)&gpu_vmax, sizeof(dtype));
}

void CUDA_solver::pre_process(Fields &field, Grid &grid, Discretization &discretization, dtype cpu_dt) {

    hipMemcpy(gpu_geometry_data, grid.get_geometry_data().data(), domain_size * sizeof(int), hipMemcpyHostToDevice);

    bool energycheck = field.isHeatTransfer();
    hipMemcpy(gpu_isHeatTransfer, &energycheck, sizeof(bool), hipMemcpyHostToDevice);

    if (energycheck)
        hipMemcpy(gpu_T, field.t_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_U, field.u_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_V, field.v_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_P, field.p_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);

    hipMemcpy(gpu_csrValA, field.csrValA_matrix().data(), nnzA * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_csrRowPtrA, field.csrRowPtrA_matrix().data(), (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_csrColIndA, field.csrColIndA_matrix().data(), nnzA * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(gpu_F, field.f_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_G, field.g_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_RS, field.rs_matrix().data(), grid_size * sizeof(dtype), hipMemcpyHostToDevice);
    hipMemset(d_mutex, 0, sizeof(int));

    dtype var = grid.dx();
    cpu_dx = var;
    hipMemcpy(gpu_dx, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = grid.dy();
    cpu_dy = var;
    hipMemcpy(gpu_dy, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = cpu_dt;
    hipMemcpy(gpu_dt, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_alpha();
    cpu_alpha = var;
    hipMemcpy(gpu_alpha, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_beta();
    hipMemcpy(gpu_beta, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = discretization.get_gamma();
    hipMemcpy(gpu_gamma, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_nu();
    cpu_nu = var;
    hipMemcpy(gpu_nu, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_tau();
    cpu_tau = var;
    hipMemcpy(gpu_tau, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_gx();
    hipMemcpy(gpu_gx, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = field.get_gy();
    hipMemcpy(gpu_gy, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = 0;
    hipMemcpy(gpu_umax, &var, sizeof(dtype), hipMemcpyHostToDevice);

    var = 0;
    hipMemcpy(gpu_vmax, &var, sizeof(dtype), hipMemcpyHostToDevice);

    hipMemcpy(gpu_size_x, &grid_size_x, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_size_y, &grid_size_y, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_fluid_cells_size, &grid_fluid_cells_size, sizeof(int), hipMemcpyHostToDevice);

    int var1 = GEOMETRY_PGM::moving_wall_id;
    hipMemcpy(gpu_moving_wall_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::fixed_wall_id;
    hipMemcpy(gpu_fixed_wall_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::inflow_id;
    hipMemcpy(gpu_inflow_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::outflow_id;
    hipMemcpy(gpu_outflow_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::hot_id;
    hipMemcpy(gpu_hot_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::cold_id;
    hipMemcpy(gpu_cold_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = GEOMETRY_PGM::adiabatic_id;
    hipMemcpy(gpu_adiabatic_id, &(var1), sizeof(int), hipMemcpyHostToDevice);
    var1 = grid.imax() + 2;
    hipMemcpy(gpu_n, &(var1), sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(gpu_nnzA, &grid_fluid_cells_size, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(gpu_wall_temp_a, &wall_temp_a, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_wall_temp_h, &wall_temp_h, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_wall_temp_c, &wall_temp_c, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_omega, &omg, sizeof(dtype), hipMemcpyHostToDevice);

    var = LidDrivenCavity::wall_velocity;
    hipMemcpy(gpu_wall_velocity, &var, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_UIN, &UIN, sizeof(dtype), hipMemcpyHostToDevice);
    hipMemcpy(gpu_VIN, &VIN, sizeof(dtype), hipMemcpyHostToDevice);
    var = GEOMETRY_PGM::POUT;
    hipMemcpy(gpu_POUT, &var, sizeof(dtype), hipMemcpyHostToDevice);
}

void CUDA_solver::apply_boundary() {

    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);

    FixedWallBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_T, gpu_geometry_data, gpu_fluid_id, gpu_moving_wall_id, gpu_fixed_wall_id, gpu_inflow_id, gpu_outflow_id, gpu_adiabatic_id, gpu_hot_id, gpu_cold_id, gpu_wall_temp_a, gpu_wall_temp_h, gpu_wall_temp_c, gpu_isHeatTransfer, gpu_size_x, gpu_size_y);

    MovingWallBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_moving_wall_id, gpu_wall_velocity, gpu_size_x, gpu_size_y, gpu_geometry_data);

    InFlowBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_inflow_id, gpu_UIN, gpu_VIN, gpu_size_x, gpu_size_y, gpu_geometry_data);

    OutFlowBoundary<<<num_blocks_2d, block_size_2d>>>(gpu_U, gpu_V, gpu_P, gpu_outflow_id, gpu_POUT, gpu_size_x, gpu_size_y, gpu_geometry_data);

}

void CUDA_solver::calc_pressure(int max_iter, dtype tolerance, dtype t, dtype dt) {

    dtype res = 0.;
    int iter = 0;
    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);

    do{
        apply_boundary();
        solve_pressure_red_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_P, gpu_fluid_id,gpu_dx, gpu_dy, gpu_omega, gpu_coeff, gpu_size_x, gpu_size_y, gpu_geometry_data);
        solve_pressure_black_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_P,gpu_fluid_id, gpu_dx, gpu_dy, gpu_omega, gpu_coeff, gpu_size_x, gpu_size_y, gpu_geometry_data);
        calc_res_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_P, gpu_fluid_id, gpu_dx, gpu_dy, gpu_res, gpu_size_x, gpu_size_y, gpu_fluid_cells_size, gpu_geometry_data);
        hipMemcpy((void *)&res, gpu_res, sizeof(dtype), hipMemcpyDeviceToHost);
        iter++;
    }while(res > tolerance && iter < max_iter);

    if (iter == max_iter) {
        std::cout << "Max iteration reached at " << t<<" s \n";
    }

    std::cout << "Time = " << std::setw(12) << t << " Residual = "<< std::setw(12) << res <<
    
    " Iter = " << std::setw(8) << iter << " dt = " << std::setw(12) << dt << '\n';

}

void CUDA_solver::calc_pressure_direct_solve() {
    apply_boundary();
    std::chrono::time_point<std::chrono::system_clock> start;
    std::chrono::time_point<std::chrono::system_clock> end;
    
    start = std::chrono::system_clock::now();
    update_RS<<<1, 1>>>(gpu_RS, gpu_qr_RS, gpu_P, gpu_size_x, gpu_size_y, gpu_geometry_data, gpu_dx, gpu_dy, gpu_fluid_id);
    end = std::chrono::system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Elapsed time update RS: " << elapsed.count() << " mseconds\n" << std::endl;
    
    start = std::chrono::system_clock::now();
    solve_pressure_cusolver(gpu_csrValA, gpu_csrRowPtrA, gpu_csrColIndA, gpu_qr_RS, gpu_qr_P, n, nnzA);
    end = std::chrono::system_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Elapsed time update cusolver: " << elapsed.count() << " mseconds\n" << std::endl;
    
    reset_pressure<<<1, 1>>>(gpu_P, gpu_qr_P, gpu_fluid_id, gpu_geometry_data, gpu_size_x,gpu_size_y);
}

void CUDA_solver::calc_T() {
    hipMemcpy(gpu_T_temp, gpu_T, grid_size * sizeof(dtype), hipMemcpyDeviceToDevice);
    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);
    calc_T_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_T, gpu_T_temp, gpu_U, gpu_V, gpu_fluid_id, gpu_dx, gpu_dy, gpu_dt, gpu_alpha, gpu_gamma, gpu_size_x, gpu_size_y, gpu_geometry_data);
}

void CUDA_solver::calc_fluxes() {
    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);
    calc_fluxes_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_F,gpu_G,gpu_U,gpu_V,gpu_T, gpu_fluid_id, gpu_geometry_data,gpu_gx,gpu_gy,gpu_dx,gpu_dy,gpu_size_x, gpu_size_y, gpu_gamma, gpu_beta, gpu_nu, gpu_dt, gpu_isHeatTransfer);
    fluxes_bc_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_F,gpu_G,gpu_U,gpu_V, gpu_fluid_id, gpu_fixed_wall_id, gpu_adiabatic_id, gpu_hot_id, gpu_cold_id, gpu_moving_wall_id, gpu_inflow_id, gpu_outflow_id, gpu_geometry_data,gpu_size_x, gpu_size_y);
}

void CUDA_solver::calc_rs() {
    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);
    calc_rs_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_RS, gpu_F,gpu_G, gpu_fluid_id, gpu_dx,gpu_dy, gpu_dt, gpu_size_x, gpu_size_y, gpu_geometry_data);
}

void CUDA_solver::calc_velocities() {
    num_blocks_2d = get_num_blocks_2d(grid_size_x, grid_size_y);
    calc_velocities_kernel<<<num_blocks_2d, block_size_2d>>>(gpu_F, gpu_G, gpu_U, gpu_V, gpu_P, gpu_fluid_id, gpu_outflow_id, gpu_dx, gpu_dy, gpu_size_x, gpu_size_y, gpu_dt, gpu_geometry_data);

}

dtype CUDA_solver::calc_dt() {

    num_blocks = get_num_blocks(grid_size);
    dtype t[4];
    dtype result;

    // // Typecasting raw pointer to thrust device pointer
    // thrust::device_ptr<dtype> thrust_U = thrust::device_pointer_cast(gpu_U);
    // thrust::device_ptr<dtype> thrust_V = thrust::device_pointer_cast(gpu_V);

    // // Finding max and min element in U and V
    // thrust::device_ptr<dtype> thrust_U_max = thrust::max_element(thrust_U, thrust_U + grid_size);
    // thrust::device_ptr<dtype> thrust_U_min = thrust::min_element(thrust_U, thrust_U + grid_size);
    // thrust::device_ptr<dtype> thrust_V_max = thrust::max_element(thrust_V, thrust_V + grid_size);
    // thrust::device_ptr<dtype> thrust_V_min = thrust::min_element(thrust_V, thrust_V + grid_size);

    // // Finding Maximum between max element and -(min element) as Abs max needed.
    // thrust::maximum<dtype> get_max;
    // dtype umax = get_max(*thrust_U_max, -(*thrust_U_min));
    // dtype vmax = get_max(*thrust_V_max, -(*thrust_V_min));


    // t[0] = 1 / (2 * (cpu_nu) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    // t[1] = (cpu_dx) / (umax);
    // t[2] = (cpu_dy) / (vmax);   
    // t[3] = 1 / (2 * (cpu_alpha) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    // dtype temp_dt =t[0];

    // Previous implementation
    max_abs_element_kernel<<<num_blocks, block_size>>>(gpu_U, gpu_size_x, gpu_size_y, d_mutex, gpu_umax);
    max_abs_element_kernel<<<num_blocks, block_size>>>(gpu_V, gpu_size_x, gpu_size_y, d_mutex, gpu_vmax);
    hipMemcpy((void *)&cpu_umax, gpu_umax, sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)&cpu_vmax, gpu_vmax, sizeof(dtype), hipMemcpyDeviceToHost);
    t[0] = 1 / (2 * (cpu_nu) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    t[1] = (cpu_dx) / (cpu_umax);
    t[2] = (cpu_dy) / (cpu_vmax);   
    t[3] = 1 / (2 * (cpu_alpha) * (1/((cpu_dx)*(cpu_dx)) + 1/((cpu_dy)*(cpu_dy))));
    dtype temp_dt =t[0];

    for(int i=1; i<4; i++)
    {
        if(t[i]<temp_dt)
            temp_dt = t[i];
    }

    result = (cpu_tau) * temp_dt;
    hipMemcpy(gpu_dt, &result, sizeof(dtype), hipMemcpyHostToDevice);
    return result;

}

void CUDA_solver::post_process(Fields &field) {
    
    if (field.isHeatTransfer())
        hipMemcpy((void *)field.t_matrix().data(), gpu_T, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    
    hipMemcpy((void *)field.f_matrix().data(), gpu_F, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.g_matrix().data(), gpu_G, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.rs_matrix().data(), gpu_RS, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.u_matrix().data(), gpu_U, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.v_matrix().data(), gpu_V, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
    hipMemcpy((void *)field.p_matrix().data(), gpu_P, grid_size * sizeof(dtype), hipMemcpyDeviceToHost);
}

CUDA_solver::~CUDA_solver() {
    hipFree(gpu_geometry_data);
    hipFree(gpu_T);
    hipFree(gpu_U);
    hipFree(gpu_V);
    hipFree(gpu_P);
    hipFree(gpu_F);
    hipFree(gpu_G);
    hipFree(gpu_RS);
    hipFree(gpu_T_temp);
    hipFree(gpu_dx);
    hipFree(gpu_dy);
    hipFree(gpu_dt);
    hipFree(gpu_gx);
    hipFree(gpu_gy);
    hipFree(gpu_nu);
    hipFree(gpu_beta);
    hipFree(gpu_gamma);
    hipFree(gpu_alpha);
    hipFree(gpu_tau);
    hipFree(gpu_size_x);
    hipFree(gpu_size_y);
    hipFree(gpu_fluid_id);
    hipFree(gpu_fixed_wall_id);
    hipFree(gpu_moving_wall_id);
    hipFree(gpu_inflow_id);
    hipFree(gpu_outflow_id);
    hipFree(gpu_adiabatic_id);
    hipFree(gpu_hot_id);
    hipFree(gpu_cold_id);
    hipFree(gpu_wall_temp_a);
    hipFree(gpu_wall_temp_c);
    hipFree(gpu_wall_temp_h);
    hipFree(gpu_isHeatTransfer);
    hipFree(gpu_UIN);
    hipFree(gpu_VIN);
    hipFree(gpu_POUT);
    hipFree(gpu_wall_velocity);
    hipFree(gpu_umax);
    hipFree(gpu_vmax);
    hipFree(gpu_omega);
    hipFree(gpu_coeff);
    hipFree(gpu_rloc);
    hipFree(gpu_val);
    hipFree(gpu_res);
    hipFree(gpu_fluid_cells_size);
    hipFree(d_mutex);
    hipFree(gpu_csrValA);
    hipFree(gpu_csrColIndA);
    hipFree(gpu_csrRowPtrA);
    hipFree(gpu_nnzA);
    hipFree(gpu_n);
    hipFree(gpu_qr_P);
    hipFree(gpu_qr_RS);
}
